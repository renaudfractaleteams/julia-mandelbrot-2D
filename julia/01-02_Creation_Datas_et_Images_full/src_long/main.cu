#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>  // Pour fprintf et stderr
#include <stdlib.h> // Pour les fonctions standard C comme malloc
#include <stdint.h>

// Kernel CUDA pour générer une image fractale
__global__ void Kernel_Picture(ParameterPicture parameter_picture, long *data)
{
    // Calcul des indices 3D pour chaque thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Obtenir l'index 2D correspondant
    long index = parameter_picture.Get_index_2D(idx, idy, idz);

    // Si l'index est valide
    if (index >= 0)
    {
        // Obtenir la position complexe correspondante
        double2 pos_double = parameter_picture.GetPose_double(idx, idy, idz);
        Complex z(pos_double.x, pos_double.y);
        Complex c(pos_double.x, pos_double.y);

        // Si le type de fractale est Julia, utiliser les coefficients de Julia
        if (parameter_picture.type_fractal == Type_Fractal::Julia)
        {
            c.x = parameter_picture.coef_julia.x;
            c.y = parameter_picture.coef_julia.y;
        }
        
        long iter = 0;

        // Calculer le nombre d'itérations pour la fractale
        while (z.norm() < 2.0 && iter < parameter_picture.iter_max)
        {
            z = z.power(parameter_picture.power_value) + c;
            iter++;
        }

        // Stocker le nombre d'itérations dans le tableau de données
        data[index] = iter;
    }
}

// Fonction pour exécuter le kernel CUDA
hipError_t RUN(ParameterPicture parameter_picture, long *datas, int id_cuda)
{
    // Calculer la taille des données à allouer
    size_t size = parameter_picture.Get_size_array_2D() * sizeof(long);
    long *dev_datas = 0;
    hipError_t cudaStatus;

    // Définir la configuration des threads et des blocs
    const dim3 threadsPerBlock(16, 16, 4);
    const dim3 numBlocks((parameter_picture.lenG + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                         (parameter_picture.lenG + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                         (parameter_picture.lenG + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // Sélectionner le GPU à utiliser
    cudaStatus = hipSetDevice(id_cuda);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allouer de la mémoire sur le GPU pour les données
    cudaStatus = hipMalloc((void **)&dev_datas, size);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Lancer le kernel CUDA
    Kernel_Picture<<<numBlocks, threadsPerBlock>>>(parameter_picture, dev_datas);

    // Vérifier si le lancement du kernel a échoué
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Kernel_Picture launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Attendre la fin de l'exécution du kernel
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel_Picture!\n", cudaStatus);
        goto Error;
    }

    // Copier les données du GPU vers la mémoire de l'hôte
    cudaStatus = hipMemcpy(datas, dev_datas, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Libérer la mémoire allouée sur le GPU
    hipFree(dev_datas);

    // Réinitialiser le GPU
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return cudaStatus;
    }

    return hipSuccess;

Error:
    // En cas d'erreur, libérer la mémoire allouée sur le GPU
    hipFree(dev_datas);
    return cudaStatus;
}
